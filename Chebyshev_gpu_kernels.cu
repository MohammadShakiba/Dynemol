
#include <hip/hip_runtime.h>
#ifdef USE_GPU

#include <cuda.h>
#include "cublas_v2.h"
#include "magma_operators.h"

// Array element correspondig to matrix element (i,j) in column major order
#define IDX( i, j, LD ) ((i) + (j)*(LD))


//-------------------
// external variables
extern cublasHandle_t myHandle;
extern cudaStream_t   cublas_default, stream[];
extern const int      nStreams;


//----------------------------------------------
// Prototypes:
__host__ void
acummulate_vec_ax_async(
    const int n, const int ld, const int k,
    const cuDoubleComplex * __restrict__ vecA,
    const cuDoubleComplex * __restrict__ vecsX,
    const cuDoubleComplex * __restrict__ vecY,
    const cudaStream_t stream );

// __global__ void
// acummulate_vec_ax_kernel(
//     const int n, const int ld, const int k,
//     const cuDoubleComplex * __restrict__ vecA,
//     const cuDoubleComplex * __restrict__ vecsX,
//     cuDoubleComplex       * __restrict__ vecY );

__global__ void
acummulate_vec_ax_kernel(
    const int n, const int ld, const int k,
    const cuDoubleComplex * __restrict__ vecsX,
    cuDoubleComplex       * __restrict__ vecY );

__host__ void
Zvec_subtract(
    const int n,
    const cuDoubleComplex * x,
    const cuDoubleComplex * y,
    cuDoubleComplex       * z,
    const cudaStream_t stream );

__global__ void
Zvec_sub_yinplace_kernel(
    const int n,
    const cuDoubleComplex * __restrict__ x,
    cuDoubleComplex       * __restrict__ y );

__global__ void
Zvec_sub_kernel(
    const int n,
    const cuDoubleComplex * __restrict__ x,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z );

__host__ void
fused_Zxpby_and_subtract(
    const int n,
    const cuDoubleComplex * x,
    const cuDoubleComplex   a,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z,
    cuDoubleComplex       * d,
    const cudaStream_t stream );

__global__ void
fused_Zxpby_and_subtract_kernel(
    const int n,
    const cuDoubleComplex * x,
    const cuDoubleComplex   a,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z,
    cuDoubleComplex       * d );

__host__ void
Zaxpby_async(
    const int n,
    const cuDoubleComplex & a,
    const cuDoubleComplex * __restrict__ x,
    const cuDoubleComplex & b,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z,
    const cudaStream_t stream );

__global__ void
Zaxpby_async_kernel(
    const int n,
    const cuDoubleComplex a,
    const cuDoubleComplex * __restrict__ x,
    const cuDoubleComplex b,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z);

__host__ void
hadamard_minus(
    const int n,
    const int m,
    const double * const __restrict__ x,
    const double * const __restrict__ y,
    double       * const __restrict__ z,
    const cudaStream_t stream );

__global__ void
hadamard_minus_kernel(
    const int n,
    const double * const __restrict__ x,
    const double * const __restrict__ y,
    double       * const __restrict__ z);

__host__ void
calculate_A(
    const int n,
    const int ld,
    const cuDoubleComplex * const __restrict__ bra,
    const cuDoubleComplex * const __restrict__ ket,
    double                * const __restrict__ A,
    const cudaStream_t stream );

__global__ void
calculate_rho_kernel(
    const int n,
    const int ld,
    const cuDoubleComplex * const __restrict__ bra,
    const cuDoubleComplex * const __restrict__ ket,
    double                * const __restrict__ rho );


//----------------------------------------------
// Functions / kernels:

//----------------------------------------------
// __constant__ version
// MAX_ORDER must have the same value it has in Fortran code
#define MAX_ORDER 25
__constant__ cuDoubleComplex cA[MAX_ORDER];

__host__ void
acummulate_vec_ax_async(
    const int n, const int ld, const int k,
    const cuDoubleComplex * __restrict__ vecA,
    const cuDoubleComplex * __restrict__ vecsX,
    cuDoubleComplex       * __restrict__ vecY,
    const cudaStream_t stream )
{  
    const int Threads = 128;                         // Threads per block ## opt. for SM >= 3.0
    const int Blocks = (n + Threads-1) / Threads;    // We need enough blocks to span all the elements
    
    cudaMemcpyToSymbolAsync( cA, vecA, k, 0, cudaMemcpyDeviceToDevice, stream );

    acummulate_vec_ax_kernel <<< Blocks, Threads, 0, stream >>> (n, ld, k, vecsX, vecY);
}

//- - - - - - - - - - - - - - - - - - - - - - - -
__global__ void
acummulate_vec_ax_kernel(
    const int n, const int ld, const int k,
//     const cuDoubleComplex * __restrict__ vecA,  // using constant memory now
    const cuDoubleComplex * __restrict__ vecsX,
    cuDoubleComplex       * __restrict__ vecY )
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if( i < n )
    {
        cuDoubleComplex res = make_cuDoubleComplex( 0.0, 0.0 );
        for( int j=0; j<k; ++j )
        {
            res += cA[j] * vecsX[i];
            vecsX += ld;
        }

        vecY[i] = res;
    }
}

// //----------------------------------------------
// // __shared__ version
// __host__ void
// acummulate_vec_ax_async(
//     const int n, const int ld, const int k,
//     const cuDoubleComplex * __restrict__ vecA,
//     const cuDoubleComplex * __restrict__ vecsX,
//     cuDoubleComplex       * __restrict__ vecY,
//     const cudaStream_t stream )
// {  
//     const int Threads = 128;                         // Threads per block ## opt. for SM >= 3.0
//     const int Blocks = (n + Threads-1) / Threads;    // We need enough blocks to span all the elements
// 
//     acummulate_vec_ax_kernel <<< Blocks, Threads, 0, stream >>> (n, ld, k, vecA, vecsX, vecY);
// }
// 
// //- - - - - - - - - - - - - - - - - - - - - - - -
// __global__ void
// acummulate_vec_ax_kernel(
//     const int n, const int ld, const int k,
//     const cuDoubleComplex * __restrict__ vecA,
//     const cuDoubleComplex * __restrict__ vecsX,
//     cuDoubleComplex       * __restrict__ vecY )
// {
//     const int i = blockIdx.x*blockDim.x + threadIdx.x;
//     __shared__ cuDoubleComplex A[MAX_ORDER];
//     
//     // load vecA into A
//     if ((threadIdx.x == 0) && (i < k))
//         A[i] = vecA[i];
//     __syncthreads();
// 
//     if( i < n )
//     {
//         cuDoubleComplex res = make_cuDoubleComplex( 0.0, 0.0 );
//         for( int j=0; j<k; ++j )
//         {
//             res += A[j] * vecsX[i];
//             vecsX += ld;
//         }
// 
//         vecY[i] = res;
//     }
// }


//----------------------------------------------
__host__ void
Zvec_subtract(
    const int n,
    const cuDoubleComplex * x,
    const cuDoubleComplex * y,
    cuDoubleComplex       * z,
    const cudaStream_t stream )
{
    const int Threads = 128;                         // Threads per block ## opt. for SM >= 3.0
    const int Blocks = (n + Threads-1) / Threads;    // We need enough blocks to span all the elements

    if( z == y )
        Zvec_sub_yinplace_kernel <<< Blocks, Threads, 0, stream >>> (n, x, z);
//     else if( z == x )
//         // not used
    else
        Zvec_sub_kernel <<< Blocks, Threads, 0, stream >>> (n, x, y, z);
}

//- - - - - - - - - - - - - - - - - - - - - - - -
__global__ void
Zvec_sub_yinplace_kernel(
    const int n,
    const cuDoubleComplex * __restrict__ x,
    cuDoubleComplex       * __restrict__ y )
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;

    if( i < n )
        y[i] = x[i] - y[i];
}

//- - - - - - - - - - - - - - - - - - - - - - - -
__global__ void
Zvec_sub_kernel(
    const int n,
    const cuDoubleComplex * __restrict__ x,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z )
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;

    if( i < n )
        z[i] = x[i] - y[i];
}


//----------------------------------------------
// z = x + a*y
// d = z - x
__host__ void
fused_Zxpby_and_subtract(
    const int n,
    const cuDoubleComplex * x,
    const cuDoubleComplex   a,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z,
    cuDoubleComplex       * d,
    const cudaStream_t stream )
{
    const int Threads = 128;                         // Threads per block ## opt. for SM >= 3.0
    const int Blocks = (n + Threads-1) / Threads;    // We need enough blocks to span all the elements

    fused_Zxpby_and_subtract_kernel <<< Blocks, Threads, 0, stream >>> ( n, x, a, y, z, d );
}

//- - - - - - - - - - - - - - - - - - - - - - - -
__global__ void
fused_Zxpby_and_subtract_kernel(
    const int n,
    const cuDoubleComplex * x,
    const cuDoubleComplex   a,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z,
    cuDoubleComplex       * d )
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;

    if( i < n )
    {
        const cuDoubleComplex xx = x[i];     // just in case that d == x
        z[i] = cuCfma( a, y[i], xx );        // z[i] = xx + a*y[i];
        d[i] = z[i] - xx;
    }
}


//----------------------------------------------
// z = a*x + b*y
__host__ void
Zaxpby_async(
    const int n,
    const cuDoubleComplex &              a,
    const cuDoubleComplex * __restrict__ x,
    const cuDoubleComplex &              b,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z,
    const cudaStream_t stream )
{
    const int Threads = 128;                                // Threads per block ## opt. for SM >= 3.0
    const int Blocks = (n + Threads-1) / Threads;           // We need enough blocks to span all the elements

    Zaxpby_async_kernel <<< Blocks, Threads, 0, stream >>> ( n, a, x, b, y, z );
}

//- - - - - - - - - - - - - - - - - - - - - - - -
__global__ void
Zaxpby_async_kernel(
    const int n,
    const cuDoubleComplex                a,
    const cuDoubleComplex * __restrict__ x,
    const cuDoubleComplex                b,
    const cuDoubleComplex * __restrict__ y,
    cuDoubleComplex       * __restrict__ z)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;

    if( i < n )  z[i] = a*x[i] + b*y[i];
}






//==============================================
// Kernels for diabatic-Ehrenfest

//----------------------------------------------
__host__ void
hadamard_minus(
    const int n,
    const int m,
    const double * const __restrict__ x,
    const double * const __restrict__ y,
    double       * const __restrict__ z,
    const cudaStream_t stream )
{
    const int N = n*m;                               // for the sake of hadamard prodruct, pretend matrices are big vectors
    const int threads = 128;                         // Threads per block ## opt. for SM >= 3.0
    const int blocks = (N + threads-1) / threads;    // We need enough blocks to span all the elements

    hadamard_minus_kernel <<< blocks, threads, 0, stream >>> (N, x, y, z);
}

//- - - - - - - - - - - - - - - - - - - - - - - -
__global__ void
hadamard_minus_kernel(
    const int n,
    const double * const __restrict__ x,
    const double * const __restrict__ y,
    double       * const __restrict__ z )
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < n)
    {
//         const double zz = z[i];
        z[i] = x[i] * y[i] - z[i];
    }
}


//- - - - - - - - - - - - - - - - - - - - - - - -
// ρ = Re{ ket(j,1)*bra(i,1) -  ket(j,2)*bra(i,2) }
__global__ void
calculate_rho_kernel(
    const int n,
    const int ld,
    const cuDoubleComplex * const __restrict__ bra,
    const cuDoubleComplex * const __restrict__ ket,
    double                * const __restrict__ rho )
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y);
    
    __shared__ cuDoubleComplex ket_j[2];
    
    if ((i < n) && (j < n))
    {
        // first thread loads ket elements into shared memory
        // all threads within the block access the same values
        if (threadIdx.x == 0)
        {
            ket_j[0] = ket[ IDX(j, 0, ld) ];
            ket_j[1] = ket[ IDX(j, 1, ld) ];
        }
        __syncthreads();   // wait for ket to be loaded
        
        rho[ IDX(i, j, ld) ] = real( ket_j[0] * bra[ IDX(i, 0, ld) ] )
                             - real( ket_j[1] * bra[ IDX(i, 1, ld) ] );
    }
    /* fortran code:
    do j = 1, N
        ket_j(:) = AO_ket(j,:)
        do i = 1, N
            rho_eh(i,j) = real( ket_j(1)*AO_bra(i,1) ) - real( ket_j(2)*AO_bra(i,2) )
        end do
    end do
    */
}


//----------------------------------------------
__host__ void
calculate_A(
    const int n,
    const int ld,
    const cuDoubleComplex * const __restrict__ bra,
    const cuDoubleComplex * const __restrict__ ket,
    double                * const __restrict__ A,
    const cudaStream_t stream )
{
    // 1) calculate  ρ = Re{ ket(j,1)*bra(i,1) -  ket(j,2)*bra(i,2) }
    // result is stored in A
    // each block works on the same column
    dim3 threads(128, 1);
    dim3 blocks( (n  + threads.x - 1)/threads.x,
                 (ld + threads.y - 1)/threads.y ); 

    calculate_rho_kernel <<< blocks, threads, 0, stream >>> (n, ld, bra, ket, A);
    
    // 2) A = (ρ + ρ^T) / 2
    double alpha = 0.5;
    cublasSetStream( myHandle, stream );
    cublasDgeam(myHandle, CUBLAS_OP_N, CUBLAS_OP_T, n, n, &alpha, A, ld, &alpha, A, ld, A, ld);
}


#endif
